#include "hip/hip_runtime.h"
/*
 * Example of how to use the mxGPUArray API in a MEX file.  This example shows
 * how to write a MEX function that takes a gpuArray input and returns a
 * gpuArray output, e.g. B=mexFunction(A).
 *
 * Copyright 2012 The MathWorks, Inc.
 */

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include <iostream>
#include <math.h>
#include <stdio.h>


//Constrains
static const int MAX_THREADS_CUDA = 1024; //adjust it for your GPU. This is correct for a 2.0 architecture
static const int MAX_BLOCKS_CUDA = 65535;
static const int dimsImage = 3;//so thing can be set at co0mpile time


void convolution3DfftCUDAFull(float* im, int* imDim, float* kernel, int* kernelDim, float * convResult);



/*
 * Host code
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
	float* im; int* imDim; float* kernel; int* kernelDim; float * convResult;

    /* Retrieve the input data */
	im = (float*)mxGetData(prhs[0]);
	imDim = (int*)mxGetData(prhs[1]);
	kernel = (float*)mxGetData(prhs[2]);
	kernelDim = (int*)mxGetData(prhs[3]);


	/* Create an mxArray for the output data */
	const mwSize dims[] = { imDim[0], imDim[1], imDim[2]};
	plhs[0] = mxCreateNumericArray(3, dims, mxSINGLE_CLASS, mxREAL);
	//Create pointer for output data
	convResult = (float*)mxGetData(plhs[0]);

	convolution3DfftCUDAFull(im, imDim, kernel, kernelDim, convResult);	

}



__device__ static const float PI_2 = 6.28318530717958620f;
__device__ static const float PI_1 = 3.14159265358979310f;

////////////////////////////////////////////////////////////////////////////////
// Modulate Fourier image of padded data by Fourier image of padded kernel
// and normalize by FFT size
////////////////////////////////////////////////////////////////////////////////
//Adapted from CUDA SDK examples
__device__ void mulAndScale(hipfftComplex& a, const hipfftComplex& b, const float& c)
{
	hipfftComplex t = { c * (a.x * b.x - a.y * b.y), c * (a.y * b.x + a.x * b.y) };
	a = t;
};

__global__ void __launch_bounds__(MAX_THREADS_CUDA)  modulateAndNormalize_kernel(hipfftComplex *d_Dst, hipfftComplex *d_Src, long long int dataSize, float c)
{
	long long int i = (long long int)blockDim.x * (long long int)blockIdx.x + (long long int)threadIdx.x;
	long long int offset = (long long int)blockDim.x * (long long int)gridDim.x;
	while (i < dataSize)
	{

		hipfftComplex a = d_Src[i];
		hipfftComplex b = d_Dst[i];

		mulAndScale(a, b, c);
		d_Dst[i] = a;

		i += offset;
	}
};

//we use nearest neighbor interpolation to access FFT coefficients in the kernel
__global__ void __launch_bounds__(MAX_THREADS_CUDA)  modulateAndNormalizeSubsampled_kernel(hipfftComplex *d_Dst, hipfftComplex *d_Src, int kernelDim_0, int kernelDim_1, int kernelDim_2, int imDim_0, int imDim_1, int imDim_2, long long int datasize, float c)
{

	float r_0 = ((float)kernelDim_0) / ((float)imDim_0); //ratio between image size and kernel size to calculate access
	float r_1 = ((float)kernelDim_1) / ((float)imDim_1);
	float r_2 = ((float)kernelDim_2) / ((float)imDim_2);

	long long int i = (long long int)blockDim.x * (long long int)blockIdx.x + (long long int)threadIdx.x;
	long long int offset = (long long int)blockDim.x * (long long int)gridDim.x;
	int k_0, k_1, k_2;
	int aux;
	float auxExp, auxSin, auxCos;
	while (i < datasize)
	{
		//for each dimension we need to access k_i*r_i  i=0, 1, 2
		aux = 1 + imDim_2 / 2;
		k_2 = i % aux;
		aux = (i - k_2) / aux;
		k_1 = aux % imDim_1;
		k_0 = (aux - k_1) / imDim_1;

		hipfftComplex b = d_Dst[i];

		//apply shift in fourier domain since we did not apply fftshift to kernel (so we could use the trick of assuming the kernel is padded with zeros and then just subsample FFT)
		/* This is how we would do it in Matlab (linear phase change)
		auxExp = k_0 * r_0;
		auxExp += k_1 * r_1;
		auxExp += k_2 * r_2;
		auxExp *= PI_1;
		auxSin = sin(auxExp);
		auxCos = cos(auxExp);
		auxExp = b.x * auxCos - b.y * auxSin;

		b.y = b.x * auxSin + b.y * auxCos;
		b.x = auxExp;
		*/

		//add the ratio to each dimension and apply nearest neighbor interpolation
		//k_2 = min((int)(r_2*(float)k_2 + 0.5f),kernelDim_2-1);//the very end points need to be interpolated as "ceiling" instead of round or we can get oout of bounds access
		//k_1 = min((int)(r_1*(float)k_1 + 0.5f),kernelDim_1-1);
		//k_0 = min((int)(r_0*(float)k_0 + 0.5f),kernelDim_0-1);
		k_2 = ((int)(r_2*(float)k_2 + 0.5f)) % kernelDim_2;//the very end points need to be interpolated as "ceiling" instead of round or we can get oout of bounds access
		k_1 = ((int)(r_1*(float)k_1 + 0.5f)) % kernelDim_1;
		k_0 = ((int)(r_0*(float)k_0 + 0.5f)) % kernelDim_0;
		//calculate new coordinate relative to kernel size
		aux = 1 + kernelDim_2 / 2;
		hipfftComplex a = d_Src[k_2 + aux *(k_1 + kernelDim_1 * k_0)];

		if ((k_0 + k_1 + k_2) % 2 == 1)//after much debugging it seems the phase shift is 0 or Pi (nothing in between). In Matlab is a nice linear change as programmed above
		{
			a.x = -a.x;
			a.y = -a.y;
		}
		mulAndScale(a, b, c);

		//__syncthreads();//this actually slows down the code by a lot (0.1 sec for 512x512x512)
		d_Dst[i] = a;

		i += offset;
	}
};

//WARNING: for cuFFT the fastest running index is z direction!!! so pos = z + imDim[2] * (y + imDim[1] * x)
__global__ void __launch_bounds__(MAX_THREADS_CUDA) fftShiftKernel(float* kernelCUDA, float* kernelPaddedCUDA, int kernelDim_0, int kernelDim_1, int kernelDim_2, int imDim_0, int imDim_1, int imDim_2)
{
	int kernelSize = kernelDim_0 * kernelDim_1 * kernelDim_2;

	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if (tid<kernelSize)
	{
		//find coordinates
		long long int x, y, z, aux;


		z = tid % kernelDim_2;
		aux = (tid - z) / kernelDim_2;
		y = aux % kernelDim_1;
		x = (aux - y) / kernelDim_1;
		/*
		x = tid % kernelDim_0;
		aux = (tid - x)/kernelDim_0;
		y = aux % kernelDim_1;
		z = (aux - y)/kernelDim_1;
		*/
		//center coordinates
		x -= kernelDim_0 / 2;
		y -= kernelDim_1 / 2;
		z -= kernelDim_2 / 2;

		//circular shift if necessary
		if (x<0) x += imDim_0;
		if (y<0) y += imDim_1;
		if (z<0) z += imDim_2;

		//calculate position in padded kernel
		aux = z + imDim_2 * (y + imDim_1 * x);

		//aux = x + imDim_0 * (y + imDim_1 * z);

		//copy value
		kernelPaddedCUDA[aux] = kernelCUDA[tid];//for the most part it should be a coalescent access in oth places
	}
}


//=====================================================================
//WARNING: for cuFFT the fastest running index is z direction!!! so pos = z + imDim[2] * (y + imDim[1] * x)
//NOTE: to avoid transferring a large padded kernel, since memcpy is a limiting factor 
float* convolution3DfftCUDA(float* im, int* imDim, float* kernel, int* kernelDim)
{
	float* convResult = NULL;
	float* imCUDA = NULL;
	float* kernelCUDA = NULL;
	float* kernelPaddedCUDA = NULL;


	hipfftHandle fftPlanFwd, fftPlanInv;


	long long int imSize = 1;
	long long int kernelSize = 1;
	for (int ii = 0; ii<dimsImage; ii++)
	{
		imSize *= (long long int) (imDim[ii]);
		kernelSize *= (long long int) (kernelDim[ii]);
	}

	long long int imSizeFFT = imSize + (long long int)(2 * imDim[0] * imDim[1]); //size of the R2C transform in cuFFTComplex

																				 //allocate memory for output result
	convResult = new float[imSize];

	//allocat ememory in GPU
	hipMalloc((void**)&(imCUDA), imSizeFFT*sizeof(float));//a little bit larger to allow in-place FFT
	hipMalloc((void**)&(kernelCUDA), (kernelSize)*sizeof(float));
	hipMalloc((void**)&(kernelPaddedCUDA), imSizeFFT*sizeof(float));


	//TODO: pad image to a power of 2 size in all dimensions (use whatever  boundary conditions you want to apply)
	//TODO: pad kernel to image size
	//TODO: pad kernel and image to xy(z/2 + 1) for in-place transform
	//NOTE: in the example for 2D convolution using FFT in the Nvidia SDK they do the padding in the GPU, but in might be pushing the memory in the GPU for large images.

	//printf("Copying memory (kernel and image) to GPU\n");
	hipMemcpy(kernelCUDA, kernel, kernelSize*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(imCUDA, im, imSize*sizeof(float), hipMemcpyHostToDevice);

	
	//apply ffshift to kernel and pad it with zeros so we can calculate convolution with FFT
	hipMemset(kernelPaddedCUDA, 0, imSizeFFT*sizeof(float));
	int numThreads = std::min((long long int)MAX_THREADS_CUDA, kernelSize);
	int numBlocks = std::min((long long int)MAX_BLOCKS_CUDA, (long long int)(kernelSize + (long long int)(numThreads - 1)) / ((long long int)numThreads));
	fftShiftKernel << <numBlocks, numThreads >> >(kernelCUDA, kernelPaddedCUDA, kernelDim[0], kernelDim[1], kernelDim[2], imDim[0], imDim[1], imDim[2]);


	//make sure GPU finishes before we launch two different streams
	hipDeviceSynchronize();

	//printf("Creating R2C & C2R FFT plans for size %i x %i x %i\n",imDim[0],imDim[1],imDim[2]);
	hipfftPlan3d(&fftPlanFwd, imDim[0], imDim[1], imDim[2], HIPFFT_R2C);
	cufftSetCompatibilityMode(fftPlanFwd,CUFFT_COMPATIBILITY_NATIVE); //for highest performance since we do not need FFTW compatibility
	hipfftPlan3d(&fftPlanInv, imDim[0], imDim[1], imDim[2], HIPFFT_C2R);
	cufftSetCompatibilityMode(fftPlanInv,CUFFT_COMPATIBILITY_NATIVE);


	//transforming convolution kernel; TODO: if I do multiple convolutions with the same kernel I could reuse the results at teh expense of using out-of place memory (and then teh layout of the data is different!!!! so imCUDAfft should also be out of place)
	//NOTE: from CUFFT manual: If idata and odata are the same, this method does an in-place transform.
	//NOTE: from CUFFT manual: inplace output data xy(z/2 + 1) with fcomplex. Therefore, in order to perform an in-place FFT, the user has to pad the input array in the last dimension to Nn2 + 1 complex elements interleaved. Note that the real-to-complex transform is implicitly forward.
	hipfftExecR2C(fftPlanFwd, imCUDA, (hipfftComplex *)imCUDA);
	//transforming image
	hipfftExecR2C(fftPlanFwd, kernelPaddedCUDA, (hipfftComplex *)kernelPaddedCUDA);


	//multiply image and kernel in fourier space (and normalize)
	//NOTE: from CUFFT manual: CUFFT performs un-normalized FFTs; that is, performing a forward FFT on an input data set followed by an inverse FFT on the resulting set yields data that is equal to the input scaled by the number of elements.
	numThreads = std::min((long long int)MAX_THREADS_CUDA, imSizeFFT / 2);//we are using complex numbers
	numBlocks = std::min((long long int)MAX_BLOCKS_CUDA, (long long int)(imSizeFFT / 2 + (long long int)(numThreads - 1)) / ((long long int)numThreads));
	modulateAndNormalize_kernel << <numBlocks, numThreads >> >((hipfftComplex *)imCUDA, (hipfftComplex *)kernelPaddedCUDA, imSizeFFT / 2, 1.0f / (float)(imSize));//last parameter is the size of the FFT

																																								//inverse FFT 
	hipfftExecC2R(fftPlanInv, (hipfftComplex *)imCUDA, imCUDA);

	//copy result to host
	hipMemcpy(convResult, imCUDA, sizeof(float)*imSize, hipMemcpyDeviceToHost);

	//release memory
	(hipfftDestroy(fftPlanInv));
	(hipfftDestroy(fftPlanFwd));
	hipFree(imCUDA);
	hipFree(kernelCUDA);
	hipFree(kernelPaddedCUDA);

	return convResult;
}


//WARNING: for cuFFT the fastest running index is z direction!!! so pos = z + imDim[2] * (y + imDim[1] * x)

void convolution3DfftCUDAFull(float* im, int* imDim, float* kernel, int* kernelDim, float * convResult) {

	int i, j, k, pos;
	float * kernelNew = (float *)malloc(kernelDim[0] * kernelDim[1] * kernelDim[2] * sizeof(float));

	//convert: send to convResult first
	for (k = 0; k<imDim[2]; k++) {
		for (j = 0; j<imDim[1]; j++) {
			for (i = 0; i<imDim[0]; i++) {
				pos = k + imDim[2] * (j + imDim[1] * i);
				convResult[pos] = im[k*imDim[0] * imDim[1] + j *imDim[0] + i];
			}
		}
	}

	//convert: kernel

	for (k = 0; k<kernelDim[2]; k++) {
		for (j = 0; j<kernelDim[1]; j++) {
			for (i = 0; i<kernelDim[0]; i++) {
				pos = k + kernelDim[2] * (j + kernelDim[1] * i);
				kernelNew[pos] = kernel[k*kernelDim[0] * kernelDim[1] + j *kernelDim[0] + i];			
			}
		}
	}
	

	//Do conv
	float* result = convolution3DfftCUDA(convResult, imDim, kernelNew, kernelDim);

	//Convert back
	for (k = 0; k<imDim[2]; k++) {
		for (j = 0; j<imDim[1]; j++) {
			for (i = 0; i<imDim[0]; i++) {
				pos = k + imDim[2] * (j + imDim[1] * i);
				convResult[k*imDim[0] * imDim[1] + j *imDim[0] + i] = result[pos];
			}
		}
	}


	free(kernelNew);
	free(result);

}

