#include "hip/hip_runtime.h"
/*
 * Example of how to use the mxGPUArray API in a MEX file.  This example shows
 * how to write a MEX function that takes a gpuArray input and returns a
 * gpuArray output, e.g. B=mexFunction(A).
 *
 * Copyright 2012 The MathWorks, Inc.
 */

#include "mex.h"
#include "gpu/mxGPUArray.h"
#include "hip/hip_runtime.h"
#include "hipfft/hipfft.h"
#include <iostream>
#include <math.h>
#include <stdio.h>
#include <cmath>
#include <algorithm>
#include <vector>
#include <numeric>

#include "hip/hip_runtime.h"
#include ""

//Constrains
static const int MAX_THREADS_CUDA = 1024; //adjust it for your GPU. This is correct for a 2.0 architecture
static const int MAX_BLOCKS_CUDA = 65535;
static const int dimsImage = 3;//so thing can be set at co0mpile time


void convolution3DfftCUDAFull(float* im, int* imDim, float* kernel, int* kernelDim, float * convResult);



/*
 * Host code
 */
void mexFunction(int nlhs, mxArray *plhs[],
                 int nrhs, mxArray const *prhs[])
{
	float* im; int* imDim; float* kernel; int* kernelDim; float * convResult;

    /* Retrieve the input data */
	im = (float*)mxGetData(prhs[0]);
	imDim = (int*)mxGetData(prhs[1]);
	kernel = (float*)mxGetData(prhs[2]);
	kernelDim = (int*)mxGetData(prhs[3]);


	/* Create an mxArray for the output data */
	const mwSize dims[] = { imDim[0], imDim[1], imDim[2]};
	plhs[0] = mxCreateNumericArray(3, dims, mxSINGLE_CLASS, mxREAL);
	//Create pointer for output data
	convResult = (float*)mxGetData(plhs[0]);

	convolution3DfftCUDAFull(im, imDim, kernel, kernelDim, convResult);	

}



__device__ static const float PI_2 = 6.28318530717958620f;
__device__ static const float PI_1 = 3.14159265358979310f;

////////////////////////////////////////////////////////////////////////////////
// Modulate Fourier image of padded data by Fourier image of padded kernel
// and normalize by FFT size
////////////////////////////////////////////////////////////////////////////////
//Adapted from CUDA SDK examples
__device__ void mulAndScale(hipfftComplex& a, const hipfftComplex& b, const float& c)
{
	hipfftComplex t = { c * (a.x * b.x - a.y * b.y), c * (a.y * b.x + a.x * b.y) };
	a = t;
};

__global__ void __launch_bounds__(MAX_THREADS_CUDA)  modulateAndNormalize_kernel(hipfftComplex *d_Dst, hipfftComplex *d_Src, long long int dataSize, float c)
{
	long long int i = (long long int)blockDim.x * (long long int)blockIdx.x + (long long int)threadIdx.x;
	long long int offset = (long long int)blockDim.x * (long long int)gridDim.x;
	while (i < dataSize)
	{

		hipfftComplex a = d_Src[i];
		hipfftComplex b = d_Dst[i];

		mulAndScale(a, b, c);
		d_Dst[i] = a;

		i += offset;
	}
};

//we use nearest neighbor interpolation to access FFT coefficients in the kernel
__global__ void __launch_bounds__(MAX_THREADS_CUDA)  modulateAndNormalizeSubsampled_kernel(hipfftComplex *d_Dst, hipfftComplex *d_Src, int kernelDim_0, int kernelDim_1, int kernelDim_2, int imDim_0, int imDim_1, int imDim_2, long long int datasize, float c)
{

	float r_0 = ((float)kernelDim_0) / ((float)imDim_0); //ratio between image size and kernel size to calculate access
	float r_1 = ((float)kernelDim_1) / ((float)imDim_1);
	float r_2 = ((float)kernelDim_2) / ((float)imDim_2);

	long long int i = (long long int)blockDim.x * (long long int)blockIdx.x + (long long int)threadIdx.x;
	long long int offset = (long long int)blockDim.x * (long long int)gridDim.x;
	int k_0, k_1, k_2;
	int aux;
	float auxExp, auxSin, auxCos;
	while (i < datasize)
	{
		//for each dimension we need to access k_i*r_i  i=0, 1, 2
		aux = 1 + imDim_2 / 2;
		k_2 = i % aux;
		aux = (i - k_2) / aux;
		k_1 = aux % imDim_1;
		k_0 = (aux - k_1) / imDim_1;

		hipfftComplex b = d_Dst[i];

		//apply shift in fourier domain since we did not apply fftshift to kernel (so we could use the trick of assuming the kernel is padded with zeros and then just subsample FFT)
		/* This is how we would do it in Matlab (linear phase change)
		auxExp = k_0 * r_0;
		auxExp += k_1 * r_1;
		auxExp += k_2 * r_2;
		auxExp *= PI_1;
		auxSin = sin(auxExp);
		auxCos = cos(auxExp);
		auxExp = b.x * auxCos - b.y * auxSin;

		b.y = b.x * auxSin + b.y * auxCos;
		b.x = auxExp;
		*/

		//add the ratio to each dimension and apply nearest neighbor interpolation
		//k_2 = min((int)(r_2*(float)k_2 + 0.5f),kernelDim_2-1);//the very end points need to be interpolated as "ceiling" instead of round or we can get oout of bounds access
		//k_1 = min((int)(r_1*(float)k_1 + 0.5f),kernelDim_1-1);
		//k_0 = min((int)(r_0*(float)k_0 + 0.5f),kernelDim_0-1);
		k_2 = ((int)(r_2*(float)k_2 + 0.5f)) % kernelDim_2;//the very end points need to be interpolated as "ceiling" instead of round or we can get oout of bounds access
		k_1 = ((int)(r_1*(float)k_1 + 0.5f)) % kernelDim_1;
		k_0 = ((int)(r_0*(float)k_0 + 0.5f)) % kernelDim_0;
		//calculate new coordinate relative to kernel size
		aux = 1 + kernelDim_2 / 2;
		hipfftComplex a = d_Src[k_2 + aux *(k_1 + kernelDim_1 * k_0)];

		if ((k_0 + k_1 + k_2) % 2 == 1)//after much debugging it seems the phase shift is 0 or Pi (nothing in between). In Matlab is a nice linear change as programmed above
		{
			a.x = -a.x;
			a.y = -a.y;
		}
		mulAndScale(a, b, c);

		//__syncthreads();//this actually slows down the code by a lot (0.1 sec for 512x512x512)
		d_Dst[i] = a;

		i += offset;
	}
};

//WARNING: for cuFFT the fastest running index is z direction!!! so pos = z + imDim[2] * (y + imDim[1] * x)
__global__ void __launch_bounds__(MAX_THREADS_CUDA) fftShiftKernel(float* kernelCUDA, float* kernelPaddedCUDA, int kernelDim_0, int kernelDim_1, int kernelDim_2, int imDim_0, int imDim_1, int imDim_2)
{
	int kernelSize = kernelDim_0 * kernelDim_1 * kernelDim_2;

	int tid = blockDim.x * blockIdx.x + threadIdx.x;

	if (tid<kernelSize)
	{
		//find coordinates
		long long int x, y, z, aux;


		z = tid % kernelDim_2;
		aux = (tid - z) / kernelDim_2;
		y = aux % kernelDim_1;
		x = (aux - y) / kernelDim_1;
		/*
		x = tid % kernelDim_0;
		aux = (tid - x)/kernelDim_0;
		y = aux % kernelDim_1;
		z = (aux - y)/kernelDim_1;
		*/
		//center coordinates
		x -= kernelDim_0 / 2;
		y -= kernelDim_1 / 2;
		z -= kernelDim_2 / 2;

		//circular shift if necessary
		if (x<0) x += imDim_0;
		if (y<0) y += imDim_1;
		if (z<0) z += imDim_2;

		//calculate position in padded kernel
		aux = z + imDim_2 * (y + imDim_1 * x);

		//aux = x + imDim_0 * (y + imDim_1 * z);

		//copy value
		kernelPaddedCUDA[aux] = kernelCUDA[tid];//for the most part it should be a coalescent access in oth places
	}
}




//=====================================================================
//WARNING: for cuFFT the fastest running index is z direction!!! so pos = z + imDim[2] * (y + imDim[1] * x)
//NOTE: to avoid transferring a large padded kernel, since memcpy is a limiting factor 
float* convolution3DfftCUDA(float* im, int* imDim, float* kernel, int* kernelDim)
{
	float* convResult = NULL;

	hipfftComplex* imCUDA = NULL;
	float* kernelCUDA = NULL;	//kernelSize
	float* shifted_kernel = NULL;	//imSize
	hipfftComplex* kernelPaddedCUDA = NULL;	//complexSize


	hipfftHandle fftPlanFwd, fftPlanInv;


	long long int imSize = 1;
	long long int kernelSize = 1;
	long long int complexSize = 1;

	int complexDim[3];
	complexDim[0] = (imDim[0]/2)+1;
	complexDim[1] = imDim[1];
	complexDim[2] = imDim[2];

	for (int ii = 0; ii<dimsImage; ii++)
	{
		imSize *= (long long int) (imDim[ii]);
		kernelSize *= (long long int) (kernelDim[ii]);
		complexSize *=(long long int) (complexDim[ii]);
	}
	complexSize = complexSize * 2;



	/**************** Kernel Shift***************************/
	hipMalloc((void**)&(kernelCUDA), (kernelSize)*sizeof(float));
	hipMemcpy(kernelCUDA, kernel, kernelSize*sizeof(float), hipMemcpyHostToDevice);
	hipMalloc((void**)&(shifted_kernel), imSize*sizeof(float));
	hipMemset(shifted_kernel, 0, imSize*sizeof(float));
	//apply ffshift to kernel and pad it with zeros so we can calculate convolution with FFT
	int numThreads = std::min((long long int)MAX_THREADS_CUDA, kernelSize);
	int numBlocks = std::min((long long int)MAX_BLOCKS_CUDA, (long long int)(kernelSize + (long long int)(numThreads - 1)) / ((long long int)numThreads));
	fftShiftKernel << <numBlocks, numThreads >> >(kernelCUDA, shifted_kernel, kernelDim[0], kernelDim[1], kernelDim[2], imDim[0], imDim[1], imDim[2]);

	hipFree(kernelCUDA);
	hipDeviceSynchronize();


	/**************** Kernel Pad***************************/
	hipMalloc( (void**)&(kernelPaddedCUDA), complexSize*sizeof(float) ) ;
	hipMemset( kernelPaddedCUDA, 0, complexSize*sizeof(float) );
	float* d_src = 0;
	hipfftComplex* d_dst = 0;
	for(size_t z = 0;z<imDim[2];++z){
		for(size_t y = 0;y<imDim[1];++y){
			size_t dst_line_offset = (z*complexDim[1]*complexDim[0])+ (y*complexDim[0]);
			d_dst = kernelPaddedCUDA+dst_line_offset;
			size_t src_line_offset = (z*imDim[1]*imDim[0])+ (y*imDim[0]);
			d_src = shifted_kernel + src_line_offset;
			hipMemcpy( d_dst,d_src,imDim[0]*sizeof(float),hipMemcpyDeviceToDevice);
		}
	}
	hipDeviceSynchronize();
	hipFree(shifted_kernel);

	/**************** Image Pad***************************/
	hipMalloc((void**)&(imCUDA), complexSize*sizeof(float));
	std::vector<hipfftComplex> padded_image(complexSize/2);

	float* src_begin = 0;
	hipfftComplex* dst_begin = 0;

	for(size_t z = 0;z<imDim[2];++z){
		for(size_t y = 0;y<imDim[1];++y){
			size_t dst_line_offset = (z*complexDim[1]*complexDim[0])+ (y*complexDim[0]);
			dst_begin = &padded_image[0]+(dst_line_offset);

			size_t src_line_offset = (z*imDim[1]*imDim[0])+ (y*imDim[0]);
			src_begin = im + src_line_offset;

			std::copy(src_begin,src_begin + imDim[0],(float*)dst_begin);
		}
	}
	hipMemcpy(imCUDA ,&padded_image[0],complexSize*sizeof(float),hipMemcpyHostToDevice ) ;


	//printf("Creating R2C & C2R FFT plans for size %i x %i x %i\n",imDim[0],imDim[1],imDim[2]);
	hipfftPlan3d(&fftPlanFwd, imDim[0], imDim[1], imDim[2], HIPFFT_R2C);
	//hipfftPlan3d(&fftPlanFwd, imDim[2], imDim[1], imDim[0], HIPFFT_R2C);	

	//transforming convolution kernel; TODO: if I do multiple convolutions with the same kernel I could reuse the results at teh expense of using out-of place memory (and then teh layout of the data is different!!!! so imCUDAfft should also be out of place)
	//NOTE: from CUFFT manual: If idata and odata are the same, this method does an in-place transform.
	//NOTE: from CUFFT manual: inplace output data xy(z/2 + 1) with fcomplex. Therefore, in order to perform an in-place FFT, the user has to pad the input array in the last dimension to Nn2 + 1 complex elements interleaved. Note that the real-to-complex transform is implicitly forward.
	hipfftExecR2C(fftPlanFwd, (hipfftReal *)imCUDA, (hipfftComplex *)imCUDA);
	//transforming image
	hipfftExecR2C(fftPlanFwd, (hipfftReal *)kernelPaddedCUDA, (hipfftComplex *)kernelPaddedCUDA);


	//multiply image and kernel in fourier space (and normalize)
	//NOTE: from CUFFT manual: CUFFT performs un-normalized FFTs; that is, performing a forward FFT on an input data set followed by an inverse FFT on the resulting set yields data that is equal to the input scaled by the number of elements.
	numThreads = std::min((long long int)MAX_THREADS_CUDA, complexSize / 2);//we are using complex numbers
	numBlocks = std::min((long long int)MAX_BLOCKS_CUDA, (long long int)(complexSize / 2 + (long long int)(numThreads - 1)) / ((long long int)numThreads));
	modulateAndNormalize_kernel << <numBlocks, numThreads >> >((hipfftComplex *)imCUDA, (hipfftComplex *)kernelPaddedCUDA, complexSize / 2, 1.0f / (float)(imSize));//last parameter is the size of the FFT

	hipfftDestroy(fftPlanFwd);
	hipFree( kernelPaddedCUDA);


	hipfftPlan3d(&fftPlanInv, imDim[0], imDim[1], imDim[2], HIPFFT_C2R);
	//hipfftPlan3d(&fftPlanInv, imDim[2], imDim[1], imDim[0], HIPFFT_C2R);

	//inverse FFT 
	hipfftExecC2R(fftPlanInv, (hipfftComplex *)imCUDA, (hipfftReal *)imCUDA);

	//copy result to host
	hipMemcpy(&padded_image[0], imCUDA, sizeof(float)*complexSize, hipMemcpyDeviceToHost);

	//release memory
	(hipfftDestroy(fftPlanInv));
	hipFree(imCUDA);




	float* complex_begin = 0;
	float* real_begin = 0;
	convResult = new float[imSize];

	//get the right pixel lines again
	for(size_t z = 0;z<imDim[2];++z){
		for(size_t y = 0;y<imDim[1];++y){
			size_t dst_line_offset = (z*imDim[1]*imDim[0])+ (y*imDim[0]);
			real_begin = convResult +dst_line_offset;

			size_t src_line_offset = (z*complexDim[1]*complexDim[0])+ (y*complexDim[0]);
			complex_begin = (float*)(&padded_image[0] + (src_line_offset));

			std::copy(complex_begin,complex_begin + imDim[0],real_begin);

		}
	}

		return convResult;
}




//WARNING: for cuFFT the fastest running index is z direction!!! so pos = z + imDim[2] * (y + imDim[1] * x)

void convolution3DfftCUDAFull(float* im, int* imDim, float* kernel, int* kernelDim, float * convResult) {

	int i, j, k, pos;
	float * kernelNew = (float *)malloc(kernelDim[0] * kernelDim[1] * kernelDim[2] * sizeof(float));

	//convert: send to convResult first
	for (k = 0; k<imDim[2]; k++) {
		for (j = 0; j<imDim[1]; j++) {
			for (i = 0; i<imDim[0]; i++) {
				pos = k + imDim[2] * (j + imDim[1] * i);
				convResult[pos] = im[k*imDim[0] * imDim[1] + j *imDim[0] + i];
			}
		}
	}

	//convert: kernel

	for (k = 0; k<kernelDim[2]; k++) {
		for (j = 0; j<kernelDim[1]; j++) {
			for (i = 0; i<kernelDim[0]; i++) {
				pos = k + kernelDim[2] * (j + kernelDim[1] * i);
				kernelNew[pos] = kernel[k*kernelDim[0] * kernelDim[1] + j *kernelDim[0] + i];			
			}
		}
	}
	

	//Do conv
	float* result = convolution3DfftCUDA(convResult, imDim, kernelNew, kernelDim);

	//Convert back
	for (k = 0; k<imDim[2]; k++) {
		for (j = 0; j<imDim[1]; j++) {
			for (i = 0; i<imDim[0]; i++) {
				pos = k + imDim[2] * (j + imDim[1] * i);
				convResult[k*imDim[0] * imDim[1] + j *imDim[0] + i] = result[pos];
			}
		}
	}


	free(kernelNew);
	free(result);

}

